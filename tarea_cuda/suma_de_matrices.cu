#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <armadillo>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void suma(float *A, float *B, float *C, int N){
    // Las matrices se recorren con la ordenacion de Fortran
    int j = blockIdx.x * blockDim.x + threadIdx.x; // indice filas
    int i = blockIdx.y * blockDim.y + threadIdx.y; // indice columnas
    int tid = (i * N) + j;

    if(i < N && j < N)
        C[tid] = A[tid] + B[tid];
}

void fill(float *h_A, float *h_B, float *h_C, int N){
    for(int i = 0; i<N; i++){
        int row = N*i;
        for(int j = 0; j<N; j++){
            h_A[row+j] = row+j+1;
            h_B[row+j] =  row+j+2;
            h_C[row+j] = 0;
        }
    }
}

void parallel_matrix_sum(float *h_A, float *h_B, float *h_C, int N, int numBloques, int numThreadsBloque){
    unsigned int sizeMatrix = N * N;
    unsigned int memSize = sizeMatrix * sizeof(float);
    float * d_A ,* d_B,* d_C;

    hipMalloc((void **)&d_A, memSize);
    hipMalloc((void **)&d_B, memSize);
    hipMalloc((void **)&d_C, memSize);


    hipMemcpy(d_A, h_A, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, memSize, hipMemcpyHostToDevice);


    cout << "GPU: Calculando..." << endl;
    dim3 threadsPerBlock(numThreadsBloque,numThreadsBloque);
    dim3 numBlocks (numBloques, numThreadsBloque);
    suma<<<numBlocks, threadsPerBlock>>> (d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, memSize, hipMemcpyDeviceToHost);


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


int main(void)
{
   float *h_A,*h_B,*h_C;
   int N=1000;
   size_t size=N*sizeof(float);
   h_A = (float *) malloc (size);
   h_B = (float *) malloc (size);
   h_C = (float *) malloc (size);
   fill(*h_A, *h_B,*h_C, int N);
   int numBloques=8;
   int numThreadsBloque=N/numBloques + (N%numBloques ==0 ? 0:1);
   parallel_matrix_sum(*h_A, *h_B, *h_C, N,  numBloques,  numThreadsBloque)
   return(0);



}