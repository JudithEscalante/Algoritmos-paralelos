
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#define BLUR_SIZE 1
using namespace std;

__global__
void Kernel_bor(float * in, float * out, int w, int h)
{
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  if (Col < w && Row < h)
  {
    int pixVal = 0;
    int pixels = 0;
    
    for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow)
    {
      for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol)
      {
        int curRow = Row + blurRow;
        int curCol = Col + blurCol;
        if(curRow > -1 && curRow < h && curCol > -1 && curCol < w)
        {
          pixVal += in[curRow * w + curCol];
          pixels++; 
        }
      }
    }
   
    out[Row * w + Col] = (float)(pixVal / pixels);
  }
}

void guardar(float r[225][225], float g[225][225], float b[225][225])
{
  ofstream archivo("toy_bor.dat");
  for (int i = 0; i < 225; ++i)
  {
    for (int j = 0; j < 225; ++j)
    {
          archivo<<r[i][j]<<" "<<g[i][j]<<" "<<b[i][j]<<" ";
    }
    archivo<<endl;
  }
}

void borroso(float r[225][225], float g[225][225], float b[225][225], int width, int height)
{
  float o_r[225][225];
  float o_g[225][225];
  float o_b[225][225];

  int size = width * height;
  int memSize = size * sizeof(float);

  float *d_A, *d_B;

  hipMalloc((void **) &d_A, memSize);
  hipMalloc((void **) &d_B, memSize);
  hipMemcpy(d_A, r, memSize, hipMemcpyHostToDevice);


  dim3 DimGrid(floor((width-1)/16 + 1), floor((height-1)/16+1), 1);
  dim3 DimBlock(16, 16, 1);
  Kernel_bor<<<DimGrid,DimBlock>>>(d_A, d_B, width, height);

  hipMemcpy(o_r, d_B, memSize, hipMemcpyDeviceToHost);
  Kernel_bor<<<DimGrid,DimBlock>>>(d_A, d_B, width, height);
  hipMemcpy(o_g, d_B, memSize, hipMemcpyDeviceToHost);
  hipMemcpy(d_A, b, memSize, hipMemcpyHostToDevice);

  Kernel_bor<<<DimGrid,DimBlock>>>(d_A, d_B, width, height);
  
  hipMemcpy(o_b, d_B, memSize, hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  guardar(o_r,o_g,o_b);
}

void llenar(const char *file, float r[225][225], float g[225][225], float b[225][225])
{
  char buffer[100];
  ifstream archivo2(file);
  for (int ii = 0; ii < 225; ++ii)
  {
    for (int jj = 0; jj < 225; ++jj)
    {
          archivo2>>r[ii][jj]>>g[ii][jj]>>b[ii][jj];
    }
    archivo2.getline(buffer,100);
  }
}

int main()
{
  int width=225, height=225;
  float r[225][225];
  float g[225][225];
  float b[225][225];
  llenar("toy.dat",r,g,b);
  borroso(r,g,b,width,height);
  return EXIT_SUCCESS;
}