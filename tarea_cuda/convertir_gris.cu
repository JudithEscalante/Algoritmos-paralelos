
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#define CHANNELS 3 
using namespace std;

__global__
void convertir_grises(float * Pout, float * Pin, int width, int height)
{
  int Col = threadIdx.x + blockIdx.x * blockDim.x;
  int Row = threadIdx.y + blockIdx.y * blockDim.y;
  if (Col < width && Row < height)
  {
   
    int greyOffset = Row*width + Col;
    int rgbOffset = greyOffset*CHANNELS;
    float r = Pin[rgbOffset]; 
    float g = Pin[rgbOffset + 1];
    float b = Pin[rgbOffset + 2]; 
    Pout[greyOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
  }
}

void guardar(float o[225][225])
{
  ofstream archivo("toy_gray.dat");
  for (int i = 0; i < 225; ++i)
  {
    for (int j = 0; j < 225; ++j)
    {
          archivo<<o[i][j]<<" ";
    }
    archivo<<endl;
  }
}
void llenar(const char *file, float m[225][225*3])
{
  char buffer[100];
  ifstream archivo2("toy.dat");
  for (int ii = 0; ii < 225; ++ii)
  {
    for (int jj = 0; jj < 225; ++jj)
    {
          archivo2>>m[ii][jj*3]>>m[ii][jj*3+1]>>m[ii][jj*3+2];
    }
    archivo2.getline(buffer,100);
  }
}

void Escala(float m[225][225*3],int width, int height)
{
  float o[225][225];

  int size_in = width * (height*3);
  int size_out = width * height;
  int memSize_in = size_in * sizeof(float);
  int memSize_out = size_out * sizeof(float);

  float *d_A, *d_B;

  hipMalloc((void **) &d_A, memSize_in);
  hipMalloc((void **) &d_B, memSize_out);
  hipMemcpy(d_A, m, memSize_in, hipMemcpyHostToDevice);

 
  dim3 DimGrid(floor((width-1)/16 + 1), floor((height-1)/16+1), 1);
  dim3 DimBlock(16, 16, 1);
  convertir_grises<<<DimGrid,DimBlock>>>(d_B, d_A, width, height);
  hipMemcpy(o, d_B, memSize_out, hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  guardar(o);
}




int main()
{
  int width=225, height=225;
  float m[225][225*3];
  llenar("toy.dat",m);
  Escala(m,width,height);
  return EXIT_SUCCESS;
}