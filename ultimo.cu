#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define TILED 16


__global__ void matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m)
    {
        for(int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}


__global__ void matrix_mult_tiled(int *d_a, int *d_b, int *d_result, int n)
{
    __shared__ int tile_a[TILED][TILED];
    __shared__ int tile_b[TILED][TILED];

    int row = blockIdx.y * TILED + threadIdx.y;
    int col = blockIdx.x * TILED + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        idx = row * n + sub * TILED + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by TILED
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * TILED + threadIdx.y) * n + col;
        if(idx >= n*n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < TILED; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}




void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h)
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}


int main(int argc, char const *argv[])
{
    int m, n, k;

    srand(3333);
    printf("Ingrese dimensiones :  m n and k\n");
    scanf("%d %d %d", &m, &n, &k);


    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m*n);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*k);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*k);
    hipHostMalloc((void **) &h_cc, sizeof(int)*m*k);


    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }


    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 1024;
        }
    }

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start, 0);

    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*k);
    hipMalloc((void **) &d_c, sizeof(int)*m*k);


    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + TILED - 1) / TILED;
    unsigned int grid_cols = (k + TILED - 1) / TILED;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(TILED, TILED);



    //matrix_mult_tiled<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);

    matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);


    hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);


    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Tiempo en  %dx%d . %dx%d con tiled: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);


    hipEventRecord(start, 0);

    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("Tiempo %dx%d . %dx%d en CPU: %f ms.\n\n", m, n, n, k, cpu_elapsed_time_ms);

    //printf("all results are correct!!!, speedup = %f\n", cpu_elapsed_time_ms );

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}